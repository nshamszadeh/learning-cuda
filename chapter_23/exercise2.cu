#include "hip/hip_runtime.h"
/**
Modify example 23.1 to add all the error checking as described in the chapter. 
Try modifying the code to generate errors to verify the error checking is working.
For example, try commenting out line 62 and see what happens.
 */
 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime_api.h>
 #include "../error_check_cuda.h"
 #include "../cudaDmy.cuh"
 
 #ifndef M_PI
 #define MPI 3.1415926535897932384626433832795
 #endif
 /**
  * @brief void diff(double* u, int* N, double* dx, double* du)
  *        Compute the central difference operator on periodic data
  * @param double* u: Function data, assumed periodic
  * @param int* N: Pointer to the length of the data array
  * @param double* dx: Pointer to the space step size
  * @return doulbe* dx: Pointer to the central difference of the u data
  */
 __global__ void diff(double* u, int* N, double* dx, double* du) {
     // blockIdx is a CUDA provided constant that tells the block index within the grid
     int tid = blockIdx.x;
     // notice there's no loop, each core will perform its operation on 
     // its own entry, but some cores should not participate of they are outside
     // the range
     if (tid < *N) {
         int ip = (tid + 1) % *N;
         int im = (tid + *N - 1) % *N;
         du[tid] = (u[ip] - u[im]) / (*dx) / 2.0;
     }
 }
 
 /**
  * @brief int main(int argc, char* argv[])
  *        Demonstrate a simple example for implementing a parallel finite difference operator
  * @param int argc: Should be 2
  * @param argv[1]: Length of the vector of data
  * @return: returns the initial data and its derivative
  */
 int main(int argc, char* argv[]) {
     int N = atoi(argv[1]); // Get length of vector from input
     // These addresses are in host memory
     double* u = (double*)malloc(N * sizeof(double)); // function data
     double* du = (double*)malloc(N * sizeof(double)); // derivative data
     // These addresses are in device memory
     double* dev_u; // function data
     double* dev_du; // derivative data
     double* dev_dx; // space step size
     int* dev_N; // array length
 
     // allocate memory on the device
     CheckError(hipMalloc((void**) &dev_u, N * sizeof(double)));
     CheckError(hipMalloc((void**) &dev_du, N * sizeof(double)));
     CheckError(hipMalloc((void**) &dev_dx, sizeof(double)));
     CheckError(hipMalloc((void**) &dev_N, sizeof(int)));
 
     // Initialize the function data on the host
     double dx = 2 * M_PI / N;
     for (int i = 0; i < N; i++) {
         u[i] = sin(i * dx);
     }
 
     // Copy the input data drom the host to the device
     CheckError(hipMemcpy(dev_dx, &dx, sizeof(double), hipMemcpyHostToDevice));
     CheckError(hipMemcpy(dev_u, u, N * sizeof(double), hipMemcpyHostToDevice));
     CheckError(hipMemcpy(dev_N, &N, sizeof(int), hipMemcpyHostToDevice));
 
     // Execute the finite difference kernel using N blocks
     diff<<<N, 1>>>(dev_u, dev_N, dev_dx, dev_du);
 
     // Copy the result from the device back to the host
     CheckError(hipMemcpy(du, dev_du, N * sizeof(double), hipMemcpyDeviceToHost));
     for (int i = 0; i < N; i++) {
         printf("%lf\t%lf\n", u[i], du[i]);
     }
     // Clean up all the allocated memory
     CheckError(hipFree(dev_du));
     CheckError(hipFree(dev_dx));
     CheckError(hipFree(dev_N));
     CheckError(hipFree(dev_u));
     free(u);
     free(du);
     return 0;
 }
 
 